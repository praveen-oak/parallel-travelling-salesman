#include "hip/hip_runtime.h"
#include "file_io.c"
#include "utils.c"
#include <assert.h>

//comment
#define index(i,j,cities)((i*cities)+j)



void tsp(float *distance, unsigned int cities);
int main(int argc, char * argv[])
{
	if(argc != 4){
		fprintf(stderr, "usage: tsp cities city_distance_file optimum_tour_file\n");
	}
	unsigned int cities = (unsigned int) atoi(argv[1]);

	unsigned int distance_array_size = cities*cities*sizeof(float);
	FILE *fp=fopen(argv[2], "r");
	FILE *fp_optimum=fopen(argv[3], "r");

	float *distance = (float *)malloc(distance_array_size);
	unsigned int *tour = (unsigned int *)malloc((cities+1)*sizeof(unsigned int *));
	read_files(fp, fp_optimum, distance, tour, cities);	
	tsp(distance, cities);
}


// void seq_two_opt(unsigned int *cycle, float *distance, float *cost_array, unsigned int cities){

// 	int cost_array_index;
// 	int city_i;
// 	int city_j;
// 	float temp;
// 	for(int i = 0; i < cities+1; i++){

// 		for(int j = 0; j < cities+1; j++){

// 			cost_array_index = i*(cities+1) + j;
// 			// if(i == 9 && j == 10){
// 			// 	// printf("Setting %f %d\n",cost_array[cost_array_index], cost_array_index);
// 			// 	printf("%f %d \n",distance[cycle[i]*cities + cycle[j+1]], cycle[i]*cities + cycle[j+1]);
// 			// 	printf("%f %d \n",distance[cycle[i-1]*cities + cycle[j]], cycle[i-1]*cities + cycle[j]);
// 			// 	printf("%f %d\n",distance[cycle[j]*cities + cycle[j+1]], cycle[j]*cities + cycle[j+1]);
// 			// 	printf("%f %d \n",distance[cycle[i-1]*cities + cycle[i]], cycle[i-1]*cities + cycle[i]);
// 			// }
// 			temp = cost_array[cost_array_index];
// 			temp = temp + distance[cycle[i]*cities + cycle[j+1]];
// 			temp = temp + distance[cycle[i-1]*cities + cycle[j]];
// 			temp = temp - distance[cycle[j]*cities + cycle[j+1]];
// 			temp = temp - distance[cycle[i-1]*cities + cycle[i]];
// 			// if(cost_array_index == 539){
// 			// 	printf("setting with %d %d %f %f \n",i,j,temp, cost_array[cost_array_index]);
// 			// }
// 			cost_array[cost_array_index] = temp;
// 		}
// 	}

// }




__global__ void two_opt(unsigned int *cycle, float *distance, unsigned int cities, float *min_val_array, int* min_i_array, int* min_j_array){
	float min_val = FLT_MAX;
	float temp_val;
	float min_i = -1;
	float min_j = -1;
	for(int i = blockIdx.x*blockDim.x + threadIdx.x+1; i < cities; i = i + blockDim.x*gridDim.x){
		for(int j = blockIdx.y*blockDim.y + threadIdx.y+1; j < cities; j = j + blockDim.y*gridDim.y){
			temp_val = distance[cycle[i]*cities + cycle[j+1]];
			temp_val += distance[cycle[i-1]*cities + cycle[j]];
			temp_val -= distance[cycle[j]*cities + cycle[j+1]];
			temp_val -= distance[cycle[i-1]*cities + cycle[i]];
			if(temp_val < min_val && i < j){
				min_val = temp_val;
				min_i = i;
				min_j = j;
			}
		}
	}
	int threadId = (blockIdx.x*blockDim.x + threadIdx.x)*blockDim.x*gridDim.x + (blockIdx.y*blockDim.y + threadIdx.y);
	min_val_array[threadId] = min_val;
	min_i_array[threadId] = min_i;
	min_j_array[threadId] = min_j;

}

__global__ void find_min(float *cost_array, unsigned int cities, float *min_val_array, int* min_i_array, int* min_j_array){

	int i_stride = blockDim.x*gridDim.x;
	int j_stride = blockDim.y*gridDim.y;
	int threadId = (blockIdx.x*blockDim.x + threadIdx.x)*i_stride + (blockIdx.y*blockDim.y + threadIdx.y);
	float min_val = FLT_MAX;
	float temp_val = FLT_MAX;
	int min_i;
	int min_j;
	for(int i = blockIdx.x*blockDim.x + threadIdx.x+1; i < cities; i = i + i_stride){
		for(int j = blockIdx.y*blockDim.y + threadIdx.y+1; j < cities; j = j + j_stride){
			temp_val = cost_array[i*(cities+1)+j];
			if(temp_val < min_val && i < j){
				min_i = i;
				min_j = j;
				min_val = temp_val;
			}
		}
	}
	min_val_array[threadId] = min_val;
	min_i_array[threadId] = min_i;
	min_j_array[threadId] = min_j;
}


void tsp(float *cpu_distance, unsigned int cities){

	dim3 gridDim(16, 16);
	dim3 blockDim(16, 16);
	int total_threads = blockDim.x*blockDim.y*gridDim.x*gridDim.y;

	//create and assign data to gpu distance array
	unsigned int distance_size = cities*cities*sizeof(float);
	float *gpu_distance;
	int min_index;


	CUDA_CALL(hipMalloc(&gpu_distance, distance_size));
	CUDA_CALL(hipMemcpy(gpu_distance, cpu_distance, distance_size, hipMemcpyHostToDevice));
	
	float *cpu_min_val = (float *)malloc(total_threads*sizeof(float));
	float *gpu_min_val;
	CUDA_CALL(hipMalloc(&gpu_min_val, total_threads*sizeof(float)));

	int *cpu_min_i = (int *)malloc(total_threads*sizeof(int));
	int *gpu_min_i;
	CUDA_CALL(hipMalloc(&gpu_min_i, total_threads*sizeof(int)));

	int *cpu_min_j = (int *)malloc(total_threads*sizeof(int));
	int *gpu_min_j;
	CUDA_CALL(hipMalloc(&gpu_min_j, total_threads*sizeof(int)));
	

	unsigned int cycle_size = (cities+1)*sizeof(unsigned int);
	unsigned int *cpu_cycle = (unsigned int *)malloc(cycle_size);
	unsigned int *gpu_cycle;
	CUDA_CALL(hipMalloc(&gpu_cycle, cycle_size));

	float global_minima = FLT_MAX;
	for(int i = 0; i< cities; i++){
		allocate_cycle(cpu_cycle, i, cities);

		while(true){
			float temp_cost = get_total_cost(cpu_cycle, cpu_distance, cities);
			CUDA_CALL(hipMemcpy(gpu_cycle, cpu_cycle, cycle_size, hipMemcpyHostToDevice));

			CUDA_CALL(hipMemset(gpu_min_val, 0, total_threads*sizeof(float)));
			CUDA_CALL(hipMemset(gpu_min_i,0, total_threads*sizeof(int)));
			CUDA_CALL(hipMemset(gpu_min_j, 0, total_threads*sizeof(int)));

			two_opt<<<gridDim, blockDim>>>(gpu_cycle, gpu_distance, cities, gpu_min_val, gpu_min_i, gpu_min_j);

			CUDA_CALL(hipMemcpy(cpu_min_val, gpu_min_val, total_threads*sizeof(float), hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(cpu_min_i, gpu_min_i, total_threads*sizeof(float), hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(cpu_min_j, gpu_min_j, total_threads*sizeof(float), hipMemcpyDeviceToHost));
			hipDeviceSynchronize();
			//2-opt costs have been calculated

			min_index = get_min_val(cpu_min_val,total_threads);
			if(cpu_min_val[min_index] >= -0.001){
				if(global_minima > temp_cost){
					global_minima = temp_cost;
				}
				break;
			}
			else{
				// printf("%f \n",)
				int min_i = cpu_min_i[min_index];
				int min_j = cpu_min_j[min_index];
				update_cycle(cpu_cycle, min_i, min_j);
			}
		}
	}
	printf("global minima = %f\n",global_minima);
}