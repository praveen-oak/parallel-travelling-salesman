#include "hip/hip_runtime.h"
#include "file_io.c"
#include "utils.c"
#include <assert.h>

//comment
#define index(i,j,cities)((i*cities)+j)
#define THREADS_X 16
#define THREADS_Y 16

#define BLOCKS_X 32
#define BLOCKS_Y 32
#define STREAMS 8;


struct arg_struct {
    float *cpu_distance;
    unsigned int cities;
    // unsigned int stream;
    float *return_pointer;
    hipStream_t stream;
};

void tsp(void *arguements);
int main(int argc, char * argv[])
{
	if(argc != 4){
		fprintf(stderr, "usage: tsp cities city_distance_file optimum_tour_file\n");
	}
	unsigned int cities = (unsigned int) atoi(argv[1]);

	unsigned int distance_array_size = cities*cities*sizeof(float);
	FILE *fp=fopen(argv[2], "r");
	FILE *fp_optimum=fopen(argv[3], "r");

	float *distance = (float *)malloc(distance_array_size);
	unsigned int *tour = (unsigned int *)malloc((cities+1)*sizeof(unsigned int *));
	read_files(fp, fp_optimum, distance, tour, cities);	

	struct arg_struct args[8];
	args[0].cpu_distance = distance;
	args[0].cities = cities;
	tsp(&args[0]);
}

__global__ void two_opt(unsigned int *cycle, float *distance, unsigned int cities, float *min_val_array, unsigned int* min_index_array){
	__shared__ float temp_min[THREADS_Y*THREADS_X];
	__shared__ float temp_min_index[THREADS_Y*THREADS_X];
	float min_val = FLT_MAX;
	float temp_val;
	float min_index = -1;
	for(int i = blockIdx.x*blockDim.x + threadIdx.x+1; i < cities; i = i + blockDim.x*gridDim.x){
		for(int j = blockIdx.y*blockDim.y + threadIdx.y+1; j < cities; j = j + blockDim.y*gridDim.y){
			temp_val = distance[cycle[i]*cities + cycle[j+1]];
			temp_val += distance[cycle[i-1]*cities + cycle[j]];
			temp_val -= distance[cycle[j]*cities + cycle[j+1]];
			temp_val -= distance[cycle[i-1]*cities + cycle[i]];
			if(temp_val < min_val && i < j){
				min_val = temp_val;
				min_index = i*cities+j;
			}
		}
	}


	//total threads in each block = blockDim.x*blockDim.y
	//id of thread in block = threadIdx.x*blockDim.x + threadIdx.y
	int tid =  threadIdx.x*blockDim.x + threadIdx.y;
	int bid = blockIdx.x*gridDim.x + blockIdx.y;

	temp_min[tid] = min_val;
	temp_min_index[tid] = min_index;

	for(unsigned int stride = 1; stride < blockDim.x*blockDim.y; stride*=2){
		__syncthreads();
		if(tid %(2*stride) == 0){
			if(temp_min[tid] > temp_min[tid+stride]){
				temp_min[tid] = temp_min[tid+stride];
				temp_min_index[tid] = temp_min_index[tid+stride];
			}
		}
	}
	
	min_index_array[bid] = temp_min_index[0];
	min_val_array[bid] = temp_min[0];
}

void tsp(void *arguments){

	struct arg_struct *args = (struct arg_struct *)arguments;



	float *cpu_distance = args -> cpu_distance; 
	unsigned int cities = args -> cities;
	hipStream_t stream = args -> stream;

	dim3 gridDim(BLOCKS_X, BLOCKS_Y);
	dim3 blockDim(THREADS_X, THREADS_Y);
	int min_index;
	//create and assign data to gpu distance array
	unsigned int distance_size = cities*cities*sizeof(float);
	float *gpu_distance;


	CUDA_CALL(hipMalloc(&gpu_distance, distance_size));
	CUDA_CALL(hipMemcpy(gpu_distance, cpu_distance, distance_size, hipMemcpyHostToDevice));
	


	
	float *cpu_min_val = (float *)malloc(BLOCKS_X*BLOCKS_Y*sizeof(float));
	float *gpu_min_val;
	CUDA_CALL(hipMalloc(&gpu_min_val, BLOCKS_X*BLOCKS_Y*sizeof(float)));

	unsigned int *cpu_min_index = (unsigned int *)malloc(BLOCKS_X*BLOCKS_Y*sizeof(unsigned int));
	unsigned int *gpu_min_index;
	CUDA_CALL(hipMalloc(&gpu_min_index, BLOCKS_X*BLOCKS_Y*sizeof(unsigned int)));
	

	unsigned int cycle_size = (cities+1)*sizeof(unsigned int);
	unsigned int *cpu_cycle = (unsigned int *)malloc(cycle_size);
	unsigned int *global_optimal_cycle = (unsigned int *)malloc(cycle_size);
	unsigned int *gpu_cycle;
	CUDA_CALL(hipMalloc(&gpu_cycle, cycle_size));


	// tsp_serial(cpu_distance, gpu_distance, cpu_cycle, gpu_cycle, cpu_min_val, gpu_min_val,cpu_min_index, gpu_min_index, cities, 0);

	float global_minima = FLT_MAX;
	for(int i = 0; i< 1; i++){
		allocate_cycle(cpu_cycle, i, cities);

		while(true){
			float temp_cost = get_total_cost(cpu_cycle, cpu_distance, cities);
			CUDA_CALL(hipMemcpy(gpu_cycle, cpu_cycle, cycle_size, hipMemcpyHostToDevice));
			two_opt<<<gridDim, blockDim>>>(gpu_cycle, gpu_distance, cities, gpu_min_val, gpu_min_index);

			CUDA_CALL(hipMemcpy(cpu_min_val, gpu_min_val, BLOCKS_X*BLOCKS_Y*sizeof(float), hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(cpu_min_index, gpu_min_index, BLOCKS_X*BLOCKS_Y*sizeof(float), hipMemcpyDeviceToHost));
			hipDeviceSynchronize();
			//2-opt costs have been calculated

			min_index = get_min_val(cpu_min_val,BLOCKS_X*BLOCKS_Y);
			if(cpu_min_val[min_index] >= -0.01){
				if(global_minima > temp_cost){
					global_minima = temp_cost;
					memcpy(global_optimal_cycle, cpu_cycle, cycle_size);
				}
				break;
			}
			else{
				int min_agg_index = cpu_min_index[min_index];
				update_cycle(cpu_cycle, min_agg_index/cities, min_agg_index%cities);
			}
		}
	}
	printf("global minima = %f\n",global_minima);
}