#include "hip/hip_runtime.h"
#include "file_io.c"
#include "utils.c"
#include <assert.h>
//comment
#define index(i,j,cities)((i*cities)+j)
#define THREADS_X 32
#define THREADS_Y 32

#define BLOCKS_X 16
#define BLOCKS_Y 16


__constant__ float gpu_coordinates[16000];

void tsp(float *distance, unsigned int cities);
int main(int argc, char * argv[])
{
	if(argc != 4){
		fprintf(stderr, "usage: tsp cities city_distance_file optimum_tour_file\n");
		exit(0);
	}
	unsigned int cities = (unsigned int) atoi(argv[1]);

	unsigned int distance_array_size = cities*cities*sizeof(float);
	unsigned int coordinates_size = cities*2*sizeof(float);
	FILE *fp=fopen(argv[2], "r");
	FILE *fp_optimum=fopen(argv[3], "r");

	//WARN : this variable(distance) is not being used as of now
	float *distance;
	float *co_ordinates = (float *)malloc(coordinates_size);
	unsigned int *tour = (unsigned int *)malloc((cities+1)*sizeof(unsigned int *));


	read_files(fp, fp_optimum, distance, co_ordinates, tour, cities);
	hipMemcpyToSymbol(HIP_SYMBOL(gpu_coordinates), co_ordinates, coordinates_size);
	tsp(co_ordinates, cities);
}

__global__ void two_opt(unsigned int *cycle, unsigned int cities, float *min_val_array, int* min_index_array){
	__shared__ float temp_min[THREADS_Y*THREADS_X];
	__shared__ float temp_min_index[THREADS_Y*THREADS_X];
	float min_val = FLT_MAX;
	float temp_val;
	float min_index = -1;
	for(int i = blockIdx.x*blockDim.x + threadIdx.x+1; i < cities; i = i + blockDim.x*gridDim.x){
		for(int j = blockIdx.y*blockDim.y + threadIdx.y+1; j < cities; j = j + blockDim.y*gridDim.y){
			temp_val = get_sq_root_dist(gpu_coordinates,cycle[i]*cities,cycle[j+1]);
			temp_val += get_sq_root_dist(gpu_coordinates,cycle[i-1]*cities,cycle[j]);
			temp_val -= get_sq_root_dist(gpu_coordinates,cycle[j]*cities,cycle[j+1]);
			temp_val -= get_sq_root_dist(gpu_coordinates,cycle[i-1]*cities,cycle[i]);
			if(temp_val < min_val && i < j){
				min_val = temp_val;
				min_index = i*cities+j;
			}
		}
	}


	//total threads in each block = blockDim.x*blockDim.y
	//id of thread in block = threadIdx.x*blockDim.x + threadIdx.y
	int tid =  threadIdx.x*blockDim.x + threadIdx.y;
	int bid = blockIdx.x*gridDim.x + blockIdx.y;

	temp_min[tid] = min_val;
	temp_min_index[tid] = min_index;

	for(unsigned int stride = 1; stride < blockDim.x*blockDim.y; stride*=2){
		__syncthreads();
		if(tid %(2*stride) == 0){
			if(temp_min[tid] > temp_min[tid+stride]){
				temp_min[tid] = temp_min[tid+stride];
				temp_min_index[tid] = temp_min_index[tid+stride];
			}
		}
	}
	if(tid == 0){
		min_index_array[bid] = temp_min_index[0];
		min_val_array[bid] = temp_min[0];
	}
	
}


void tsp(float *cpu_coordinates, unsigned int cities){

	dim3 gridDim(BLOCKS_X, BLOCKS_Y);
	dim3 blockDim(THREADS_X, THREADS_Y);
	int min_index;
	
	float *cpu_min_val = (float *)malloc(BLOCKS_X*BLOCKS_Y*sizeof(float));
	float *gpu_min_val;
	CUDA_CALL(hipMalloc(&gpu_min_val, BLOCKS_X*BLOCKS_Y*sizeof(float)));

	int *cpu_min_index = (int *)malloc(BLOCKS_X*BLOCKS_Y*sizeof(int));
	int *gpu_min_index;
	CUDA_CALL(hipMalloc(&gpu_min_index, BLOCKS_X*BLOCKS_Y*sizeof(int)));
	

	unsigned int cycle_size = (cities+1)*sizeof(unsigned int);
	unsigned int *cpu_cycle = (unsigned int *)malloc(cycle_size);
	unsigned int *global_optimal_cycle = (unsigned int *)malloc(cycle_size);
	unsigned int *gpu_cycle;
	CUDA_CALL(hipMalloc(&gpu_cycle, cycle_size));

	float global_minima = FLT_MAX;
	for(int i = 0; i< cities; i++){
		allocate_cycle(cpu_cycle, i, cities);

		while(true){
			float temp_cost = get_total_cost(cpu_cycle, cpu_coordinates, cities);
			CUDA_CALL(hipMemcpy(gpu_cycle, cpu_cycle, cycle_size, hipMemcpyHostToDevice));
			two_opt<<<gridDim, blockDim>>>(gpu_cycle, cities, gpu_min_val, gpu_min_index);

			CUDA_CALL(hipMemcpy(cpu_min_val, gpu_min_val, BLOCKS_X*BLOCKS_Y*sizeof(float), hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(cpu_min_index, gpu_min_index, BLOCKS_X*BLOCKS_Y*sizeof(float), hipMemcpyDeviceToHost));
			hipDeviceSynchronize();
			//2-opt costs have been calculated

			min_index = get_min_val(cpu_min_val,BLOCKS_X*BLOCKS_Y);
			if(cpu_min_val[min_index] >= -0.01){
				if(global_minima > temp_cost){
					global_minima = temp_cost;
					memcpy(global_optimal_cycle, cpu_cycle, cycle_size);
				}
				break;
			}
			else{
				int min_agg_index = cpu_min_index[min_index];
				update_cycle(cpu_cycle, min_agg_index/cities, min_agg_index%cities);
			}
		}
	}
	printf("global minima = %f\n",global_minima);
}